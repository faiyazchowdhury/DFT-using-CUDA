#include "hip/hip_runtime.h"
//
// Created by brian on 11/20/18.
//

#include "complex.h"

#include <cmath>

const float PI = 3.14159265358979f;

Complex::Complex() : real(0.0f), imag(0.0f) {}

Complex::Complex(float r) : real(r), imag(0.0f) {}

Complex::Complex(float r, float i) : real(r), imag(i) {}

Complex Complex::operator+(const Complex &b) const {
  return Complex(this->real + b.real, this->imag + b.imag);
}

Complex Complex::operator-(const Complex &b) const {
  return Complex(this->real - b.real, this->imag - b.imag);
}

Complex Complex::operator*(const Complex &b) const {
  return Complex(this->real * b.real - this->imag * b.imag,
                 this->real * b.imag + this->imag * b.real);
}

Complex Complex::mag() const {

}

Complex Complex::angle() const {

}

Complex Complex::conj() const {
  return Complex(this->real, -1 * this->imag);
}

std::ostream& operator<< (std::ostream& os, const Complex& rhs) {
    Complex c(rhs);
    // if(fabsf(rhs.imag) < 1e-6) c.imag = 0.0f;
    // if(fabsf(rhs.real) < 1e-6) c.real = 0.0f;

    if(c.imag == 0) {
        os << rhs.real;
    }
    else {
        os << "(" << rhs.real << "," << rhs.imag << ")";
    }
    return os;
}

#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <string.h>
#include "complex.h"
#include "input_image.h"
#include <thread>

using std::vector;

struct blockData
{
    int width;
    int height;
    int expOp;
    float sumOp;
};

struct ComplexCUDA
{
    float real;
    float imag;
};
__device__ struct ComplexCUDA complexMult(struct ComplexCUDA a, struct ComplexCUDA b) {
    struct ComplexCUDA c;
    c.real = a.real*b.real-a.imag*b.imag;
    c.imag = a.real*b.imag+a.imag*b.real;
    return c;
};
__device__ struct ComplexCUDA complexAdd(struct ComplexCUDA a, struct ComplexCUDA b) {
    struct ComplexCUDA c;
    c.real = a.real+b.real;
    c.imag = a.imag+b.imag;
    return c;
};
__device__ struct ComplexCUDA createComplexCUDA() {
    struct ComplexCUDA c;
    c.real = 0;
    c.imag = 0;
    return c;
};
__device__ struct ComplexCUDA createComplexCUDA(float a) {
    struct ComplexCUDA c;
    c.real = a;
    c.imag = 0;
    return c;
};
__device__ struct ComplexCUDA createComplexCUDA(float a, float b) {
    struct ComplexCUDA c;
    c.real = a;
    c.imag = b;
    return c;
};

__global__ void blockDftHoriz(struct ComplexCUDA *dftData, struct ComplexCUDA *indata, struct blockData *bd)
{
    __shared__ struct ComplexCUDA data[2048];
    data[blockIdx.x*bd->width+threadIdx.x] = indata[blockIdx.x*bd->width+threadIdx.x];
    __shared__ struct ComplexCUDA expTerm[2048];
    __shared__ struct ComplexCUDA sum[2048];

    sum[threadIdx.x] = createComplexCUDA(0);
    if (threadIdx.x+(bd->width+1)/2 < bd->width) 
    {
        sum[threadIdx.x+(bd->width+1)/2] = createComplexCUDA(0);
    }
    __syncthreads();
    for (int t = 0; t < bd->width; t++)
    {
        expTerm[threadIdx.x] = createComplexCUDA(cos(float(bd->expOp) * 2.0 * 3.14159 * float(t) * float(threadIdx.x) / float(bd->width)),sin(bd->expOp * 2.0 * 3.14159 * float(t) * float(threadIdx.x) / float(bd->width)));
        sum[threadIdx.x] = complexAdd(sum[threadIdx.x],complexMult(data[blockIdx.x*bd->width+t],expTerm[threadIdx.x]));

        if (threadIdx.x+(bd->width+1)/2 < bd->width) 
        {
            expTerm[threadIdx.x+(bd->width+1)/2] = createComplexCUDA(cos(bd->expOp * 2.0 * M_PI * float(t) * float(threadIdx.x+(bd->width+1)/2) / float(bd->width)),\
                    sin(bd->expOp * 2.0 * M_PI * float(t) * float(threadIdx.x+(bd->width+1)/2) / float(bd->width)));
            sum[threadIdx.x+(bd->width+1)/2] = complexAdd(sum[threadIdx.x+(bd->width+1)/2],complexMult(data[blockIdx.x*bd->width+t],expTerm[threadIdx.x+(bd->width+1)/2]));
        }
    }
    dftData[blockIdx.x * bd->width + threadIdx.x] = complexMult(sum[threadIdx.x],createComplexCUDA(bd->sumOp));
    if (threadIdx.x+(bd->width+1)/2 < bd->width) 
    {
        dftData[blockIdx.x * bd->width + threadIdx.x+(bd->width+1)/2] = complexMult(sum[threadIdx.x+(bd->width+1)/2],createComplexCUDA(bd->sumOp));
    }
}


__global__ void blockDftVert(struct ComplexCUDA *dftData, struct ComplexCUDA *indata, struct blockData *bd)
{
    __shared__ struct ComplexCUDA data[2048];
    data[blockIdx.x*bd->width+threadIdx.x] = indata[blockIdx.x*bd->width+threadIdx.x];
    __shared__ struct ComplexCUDA expTerm[2048];
    __shared__ struct ComplexCUDA sum[2048];

    sum[threadIdx.x] = createComplexCUDA(0);
    if (threadIdx.x+(bd->width+1)/2 < bd->width) 
    {
        sum[threadIdx.x+(bd->width+1)/2] = createComplexCUDA(0);
    }
    __syncthreads();
    for (int t = 0; t < bd->height; t++)
    {
        expTerm[threadIdx.x] = createComplexCUDA(cos(bd->expOp * 2.0 * M_PI * float(t) * float(threadIdx.x) / float(bd->height)),sin(bd->expOp * 2.0 * M_PI * float(t) * float(threadIdx.x) / float(bd->height)));
        sum[threadIdx.x] = complexAdd(sum[threadIdx.x],complexMult(data[t*bd->width+blockIdx.x],expTerm[threadIdx.x]));

        if (threadIdx.x+(bd->height+1)/2 < bd->width) 
        {
            expTerm[threadIdx.x+(bd->height+1)/2] = createComplexCUDA(cos(bd->expOp * 2.0 * M_PI * float(t) * float(threadIdx.x+(bd->height+1)/2) / float(bd->width)),\
                    sin(bd->expOp * 2.0 * M_PI * float(t) * float(threadIdx.x+(bd->height+1)/2) / float(bd->width)));
            sum[threadIdx.x+(bd->height+1)/2] = complexAdd(sum[threadIdx.x+(bd->height+1)/2],complexMult(data[t*bd->width+blockIdx.x],expTerm[threadIdx.x+(bd->height+1)/2]));
        }
    }
    dftData[threadIdx.x * bd->width + blockIdx.x] = complexMult(sum[threadIdx.x],createComplexCUDA(bd->sumOp));
    if (threadIdx.x+(bd->height+1)/2 < bd->width) 
    {
        dftData[(threadIdx.x+(bd->height+1)/2) * bd->width + blockIdx.x] = complexMult(sum[threadIdx.x+(bd->height+1)/2],createComplexCUDA(bd->sumOp));
    }
}

/**
 * Do 2d dft in one thread. If forward is false, the inverse will be done
 * (forward is the default, though)
 */
Complex *doDft(Complex *data, int width, int height, bool forward = true)
{
    // std::cout << "<doDft> VAR INIT....." ;
    int expOp = forward ? -1 : 1;
    float sumOp = forward ? float(1.0) : float(1.0 / width);
    Complex *dftData2 = new Complex[width * height];
    struct ComplexCUDA dataCUDA[width*height];
    struct ComplexCUDA dftData2CUDA[width*height];
    // struct ComplexCUDA dftDataCUDA[width*height];
    // std::cout << "done!" << std::endl;    
    
    // std::cout << "<doDft> ComplexCUDA Conversion....." ;
    for (int iH=0;iH<height;iH++)
    {
        for (int iW=0;iW<width;iW++)
        {
            dataCUDA[iH*width+iW].real = data[iH*width+iW].real;
            dataCUDA[iH*width+iW].imag = data[iH*width+iW].imag;
        }
    }
    // std::cout << "done!" << std::endl;

    // std::cout << "<doDft> CUDA VAR INIT....." ;
    struct ComplexCUDA *d_data;
    struct ComplexCUDA *d_dftData;
    struct ComplexCUDA *d_dftData2;
    struct blockData *d_bd;
    // std::cout << "done!" << std::endl;

    // std::cout << "<doDft> blockData INIT....." ;
    struct blockData bd;
    bd.width = width;
    bd.height = height;
    bd.expOp = expOp;
    bd.sumOp = sumOp;
    // std::cout << "done!" << std::endl;

    // std::cout << "<doDft> CUDA MALLOC....." ;
    hipMalloc((void **) &d_data, sizeof(dataCUDA));
    hipMalloc((void **) &d_dftData, sizeof(dataCUDA));
    hipMalloc((void **) &d_dftData2, sizeof(dataCUDA));
    hipMalloc((void **) &d_bd, sizeof(bd));
    // std::cout << "done!" << std::endl;

    // std::cout << "<doDft> CUDA MEMCPY TO DEVICE....." ;
    hipMemcpy(d_data, dataCUDA, sizeof(dataCUDA), hipMemcpyHostToDevice);
    hipMemcpy(d_bd, &bd, sizeof(bd), hipMemcpyHostToDevice);
    // std::cout << "done!" << std::endl;
    
    // std::cout << "<blockDftHoriz>....." ;
    blockDftHoriz<<<height,(width+1)/2>>>(d_dftData,d_data,d_bd);
    // blockDftHoriz<<<height,width>>>(d_dftData,d_data,d_bd);
    // std::cout << "done!" << std::endl;

    // std::cout << "<blockDftVert>....." ;
    blockDftVert<<<width,(height+1)/2>>>(d_dftData2,d_dftData,d_bd);
    // blockDftVert<<<width,height>>>(d_dftData2,d_dftData,d_bd);
    // std::cout << "done!" << std::endl;

    // std::cout << "<doDft> CUDA MEMCPY TO HOST....." ;
    // hipMemcpy(dftData2CUDA, d_dftData, sizeof(struct ComplexCUDA[width*height]), hipMemcpyDeviceToHost);
    hipMemcpy(dftData2CUDA, d_dftData2, sizeof(struct ComplexCUDA[width*height]), hipMemcpyDeviceToHost);
    // std::cout << "done!" << std::endl;

    // std::cout << "<doDft> Complex Conversion....." ;
    for (int iH=0;iH<height;iH++)
    {
        for (int iW=0;iW<width;iW++)
        {
            dftData2[iH*width+iW].real = dftData2CUDA[iH*width+iW].real;
            dftData2[iH*width+iW].imag = dftData2CUDA[iH*width+iW].imag;
        }
    }
    std::cout << "done!" << std::endl;

    // std::cout << "<doDft> CUDA MEMFREE....." ;
    hipFree(d_data);
    hipFree(d_dftData);
    hipFree(d_dftData2);
    hipFree(d_bd);
    // std::cout << "done!" << std::endl;

    return dftData2;
}

int main(int argc, char **argv)
{
    if (argc != 4)
    {
        std::cout << "wrong # inputs" << std::endl;
        return -1;
    }
    bool isForward(!strcmp(argv[1], "forward"));
    char *inputFile = argv[2];
    char *outputFile = argv[3];

    if (isForward)
    {
        std::cout << "doing forward" << std::endl;
    }
    else
    {
        std::cout << "doing reverse" << std::endl;
    }
    std::cout << inputFile << std::endl;
    InputImage im(inputFile);
    int width = im.get_width();
    int height = im.get_height();

    Complex *data = im.get_image_data();
    Complex *dftData = doDft(data, width, height, isForward);

    std::cout << "writing" << std::endl;
    im.save_image_data(outputFile, dftData, width, height);
    std::cout << "dunzo" << std::endl;
    return 0;
}
